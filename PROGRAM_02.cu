#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h> 
#include <windows.h>
#include <d2d1.h>
#include <d2d1helper.h>
#pragma comment(lib, "d2d1")
#include <mmsystem.h>
#pragma comment(lib, "winmm.lib")
#include "pegazus_main.h"
#include "PEGA_FORMAT_OBJ_NEW.h"

//***********STANDARD WIN32API ABLAKKEZELES************
HINSTANCE hInstGlob;
int SajatiCmdShow;
HWND Form1; //Ablak kezeloje
LRESULT CALLBACK WndProc0(HWND, UINT, WPARAM, LPARAM);
//******************************************************

//************************
void PEGA_drawing2D(void);
//************************************

//*********************************
//A windows program belépési pontja
//*********************************
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR szCmdLine, int iCmdShow)
{
 MSG msg;
 WNDCLASS wndclass0;
 SajatiCmdShow = iCmdShow;
 hInstGlob = hInstance;

 //*********************************
 //Ablak osztálypéldány elokészítése
 //*********************************
 wndclass0.style = CS_HREDRAW | CS_VREDRAW;
 wndclass0.lpfnWndProc = WndProc0;
 wndclass0.cbClsExtra = 0;
 wndclass0.cbWndExtra = 0;
 wndclass0.hInstance = hInstance;
 wndclass0.hIcon = LoadIcon(NULL, IDI_APPLICATION);
 wndclass0.hCursor = LoadCursor(NULL, IDC_ARROW);
 wndclass0.hbrBackground = (HBRUSH)GetStockObject(LTGRAY_BRUSH);
 wndclass0.lpszMenuName = NULL;
 wndclass0.lpszClassName = TEXT("WIN0");

 //*********************************
 //Ablak osztálypéldány regisztrációja
 //*********************************
 if (!RegisterClass(&wndclass0))
 {
  MessageBox(NULL, TEXT("Error:Program initialisation process."), TEXT("Program Start"), MB_ICONERROR);
  return 0;
 }

 //*********************************
 //Ablak létrehozása
 //*********************************
 Form1 = CreateWindow(TEXT("WIN0"),
  TEXT("CUDA - DIRECT2D"),
  (WS_OVERLAPPED | WS_SYSMENU | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_MINIMIZEBOX),
  0,
  0,
  KEPERNYO_WIDTH,
  KEPERNYO_HEIGHT,
  NULL,
  NULL,
  hInstance,
  NULL);

 //*********************************
 //Ablak megjelenítése
 //*********************************
 ShowWindow(Form1, SajatiCmdShow);
 UpdateWindow(Form1);

 //*********************************
 //Ablak üzenetkezelésének aktiválása
 //*********************************
 while (GetMessage(&msg, NULL, 0, 0))
 {
  TranslateMessage(&msg);
  DispatchMessage(&msg);
 }
 return msg.wParam;
}

//*********************************
//Az ablak callback függvénye: eseménykezelés
//*********************************
LRESULT CALLBACK WndProc0(HWND hwnd, UINT message, WPARAM wParam, LPARAM lParam)
{
 HDC hdc;
 PAINTSTRUCT ps;

 switch (message)
 {
  //*********************************
  //Ablak létrehozásakor közvetlenül
  //*********************************
 case WM_CREATE:
  /*Init*/;
  srand((unsigned)time(NULL));
  PEGA_init(hwnd);
  PEGA_create_HOST_2D_point_list(10000);
  PEGA_create_CUDA_2D_point_list(10000);
  PEGA_create_HOST_2D_line_list(10000);
  PEGA_create_CUDA_2D_line_list(10000);
  PEGA_create_HOST_2D_triangle_list(10000);
  PEGA_create_CUDA_2D_triangle_list(10000);
  return 0;
  //*********************************
  //Képernyővillogás elkerülése
  //*********************************
 case WM_ERASEBKGND:
  return (LRESULT)1;
  //*********************************
  //Ablak kliens területének újrarajzolása
  //*********************************
 case WM_PAINT:
  hdc = BeginPaint(hwnd, &ps);
  EndPaint(hwnd, &ps);
  PEGA_drawing2D();
  return 0;
  //*********************************
  //Ablak bezárása, erőforrások felszabadítása
  //*********************************
 case WM_CLOSE:
  PEGA_free_Direct2D();
  PEGA_free2D();
  DestroyWindow(hwnd);
  return 0;
  //*********************************
  //Ablak megsemmisítése
  //*********************************
 case WM_DESTROY:
  PostQuitMessage(0);
  return 0;
 }
 return DefWindowProc(hwnd, message, wParam, lParam);
}

void PEGA_drawing2D(void)
{
 int i, j;
 PEGA_clearscreen();

 // pontrajzolás
 PEGA_2D_point_reset();
 for (j = 0; j < 5000; j += 1)
  PEGA_add_2D_point(get_rnd(1899), get_rnd(999), RGB(255, 0, 0));

 // vonalrajzolás
 PEGA_2D_line_reset();
 PEGA_add_2D_line(0, 0, 1900,1000,RGB(0, 0, 0));
 PEGA_add_2D_line(1900, 0, 0, 1000, RGB(0, 0, 0));
 for (j = 0; j < 1900; j += 100)
  PEGA_add_2D_line(j, 0, j, 1000, RGB(0, 0, 0));
 PEGA_add_2D_line(0, 2, 1898, 2, RGB(0, 0, 0));
 PEGA_add_2D_line(1898, 2, 1898, 998, RGB(0, 0, 0));
 PEGA_add_2D_line(0, 998, 0, 2, RGB(0, 0, 0));
 PEGA_add_2D_line(1898, 998, 0, 998, RGB(0, 0, 0));

 // háromszögrajzolás
 PEGA_2D_triangle_reset();
 PEGA_add_2D_triangle(get_rnd(1899), get_rnd(999), get_rnd(1899), get_rnd(999), get_rnd(1899), get_rnd(999), RGB(get_rnd(255), get_rnd(255), get_rnd(255)));

 PEGA_push_points_to_GPU();
 PEGA_push_lines_to_GPU();
 PEGA_push_triangles_to_GPU();  
 PEGA_render_2D(); 
 
 PEGA_merge_down_2D_buffer();
 PEGA_swap_buffer();

 char hibauzenet[256];
 strcpy_s(hibauzenet, hipGetErrorString(hipGetLastError()));
 SetWindowTextA(Form1, hibauzenet);
}
