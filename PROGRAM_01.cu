#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h> 
#include <windows.h>
#include <d2d1.h>
#include <d2d1helper.h>
#pragma comment(lib, "d2d1")
#include <mmsystem.h>
#pragma comment(lib, "winmm.lib")

#include "pegazus_main.h"
#include "PEGA_FORMAT_OBJ_NEW.h"
#include "PEGA_FORMAT_BMP.h"

//***********STANDARD WIN32API ABLAKKEZELES************
HINSTANCE hInstGlob;
int SajatiCmdShow;
HWND Form1; //Ablak kezeloje
LRESULT CALLBACK WndProc0(HWND, UINT, WPARAM, LPARAM);
//******************************************************

//*********************************
//A windows program belépési pontja
//*********************************
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR szCmdLine, int iCmdShow)
{
 MSG msg;
 WNDCLASS wndclass0;
 SajatiCmdShow = iCmdShow;
 hInstGlob = hInstance;

 //*********************************
 //Ablak osztálypéldány előkészítése
 //*********************************
 wndclass0.style = CS_HREDRAW | CS_VREDRAW;
 wndclass0.lpfnWndProc = WndProc0;
 wndclass0.cbClsExtra = 0;
 wndclass0.cbWndExtra = 0;
 wndclass0.hInstance = hInstance;
 wndclass0.hIcon = LoadIcon(NULL, IDI_APPLICATION);
 wndclass0.hCursor = LoadCursor(NULL, IDC_ARROW);
 wndclass0.hbrBackground = (HBRUSH)GetStockObject(LTGRAY_BRUSH);
 wndclass0.lpszMenuName = NULL;
 wndclass0.lpszClassName = TEXT("WIN0");

 //*********************************
 //Ablak osztálypéldány regisztrációja
 //*********************************
 if (!RegisterClass(&wndclass0))
 {
  MessageBox(NULL, TEXT("Error:Program initialisation process."), TEXT("Program Start"), MB_ICONERROR);
  return 0;
 }

 //*********************************
 //Ablak létrehozása
 //*********************************
 Form1 = CreateWindow(TEXT("WIN0"),
  TEXT("CUDA - DIRECT2D"),
  (WS_OVERLAPPED | WS_SYSMENU | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_MINIMIZEBOX),
  0,
  0,
  KEPERNYO_WIDTH,
  KEPERNYO_HEIGHT,
  NULL,
  NULL,
  hInstance,
  NULL);

 //*********************************
 //Ablak megjelenítése
 //*********************************
 ShowWindow(Form1, SajatiCmdShow);
 UpdateWindow(Form1);

 //*********************************
 //Ablak üzenetkezelésének aktiválása
 //*********************************
 while (GetMessage(&msg, NULL, 0, 0))
 {
  TranslateMessage(&msg);
  DispatchMessage(&msg);
 }
 return msg.wParam;
}

//*********************************
//Az ablak callback függvénye: eseménykezelés
//*********************************
LRESULT CALLBACK WndProc0(HWND hwnd, UINT message, WPARAM wParam, LPARAM lParam)
{
 HDC hdc;
 PAINTSTRUCT ps;
 FILE* myfile;
 unsigned int xPos, yPos, xPos2, yPos2, fwButtons;

 switch (message)
 {
 //*********************************
 //Ablak létrehozásakor közvetlenül
 //*********************************
 case WM_CREATE:
  /*Init*/;  
  srand((unsigned)time(NULL));
  return 0;
 //*********************************
 //Képernyővillogás elkerülése
 //*********************************
 case WM_ERASEBKGND:
  return (LRESULT)1;

 //*********************************
 //Ablak kliens területének újrarajzolása
 //*********************************
 case WM_PAINT:
  hdc = BeginPaint(hwnd, &ps);
  EndPaint(hwnd, &ps);
  return 0;
 //*********************************
 //Ablak bezárása, erőforrások felszabadítása
 //*********************************
 case WM_CLOSE:
  DestroyWindow(hwnd);
  return 0;
 //*********************************
 //Ablak megsemmisítése
 //*********************************
 case WM_DESTROY:
  PostQuitMessage(0);
  return 0;
 }
 return DefWindowProc(hwnd, message, wParam, lParam);
}
